#include "hip/hip_runtime.h"
#include "headerinc.h"

#define size_seek(a) ((a) == 24 ? 54:1078)

using namespace std;

BMPHEADER *_head;
DIBHEADER *_dib;
__global__ void Rgbinv(unsigned char *a, unsigned char *b, unsigned int count)
{

int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < count)
{
	float contrast = 50;
	float correction_factor;
	correction_factor = (259.0*(contrast + 255.0)) / (255.0*(259.0 - contrast));
	float temp = ((correction_factor * ((float)a[id] - 128.0)) + 128.0); // Contract correction taking place here
	b[id] = temp >= 0 ? (temp <= 255 ? (unsigned char)temp : 255) : 0;
}
}

int main()
{
	FILE *fp, *fp1;
	unsigned char *data;
	unsigned char *d_a,*d_b;
	unsigned int colordata[256];
	fp = fopen("C:/Users/cdamo/Desktop/2.bmp", "rb");
	fp1 = fopen("C:/Users/cdamo/Desktop/a2.bmp", "wb");

	_head = (BMPHEADER *)malloc(sizeof(BMPHEADER));
	fread(_head, sizeof(BMPHEADER), 1, fp);
	if (_head->marker != 19778) {
		cout << "Not a bmp file";
	}
	else {
		_dib = (DIBHEADER *)malloc(sizeof(DIBHEADER));
		fread(_dib, sizeof(DIBHEADER), 1, fp);
		if (_dib->bits_pixel == 8) { fread(colordata, 256 * 4, 1, fp); }
		fwrite(_head, 1, sizeof(BMPHEADER), fp1);
		fwrite(_dib, 1, sizeof(DIBHEADER), fp1);
		if (_dib->bits_pixel == 8) fwrite(colordata, 1, 256 * 4, fp1);
		fseek(fp1, size_seek(_dib->bits_pixel), SEEK_SET);
		unsigned int padded = floor((float)(_dib->bits_pixel*_dib->width_pixel + 31.0)/32.0)*4 -_dib->bits_pixel/8*_dib->width_pixel ;
		DWORD pixelarray = ((_dib->bits_pixel/8*_dib->width_pixel)+padded)*_dib->height_pixel;
		data = (unsigned char *)malloc(pixelarray);
		fread(data,pixelarray,1,fp);
		if (hipMalloc(&d_a, pixelarray) != hipSuccess)
		{
		
			cout << "Nope!";
			return 0;
		}
		if (hipMalloc(&d_b, pixelarray) != hipSuccess)
		{

			cout << "Nope!";
			hipFree(d_a);
			return 0;
		}
		if (hipMemcpy(d_a, data, pixelarray, hipMemcpyHostToDevice) != hipSuccess) 
		{

			cout << "Nope!";
			hipFree(d_b);
			hipFree(d_a);
			return 0;
		}
		clock_t begin = clock();
		Rgbinv <<<pixelarray / 256 + 1 , 256>>> (d_a,d_b,pixelarray);
		clock_t end = clock();
		cout<<((double)end-begin)/CLOCKS_PER_SEC<<" Secs";
		if (hipMemcpy(data, d_b, pixelarray, hipMemcpyDeviceToHost) != hipSuccess) 
		{

			cout << "Nope!";
			delete[] data;
			hipFree(d_b);
			hipFree(d_a);
			return 0;
		}
		fwrite(data,1,pixelarray,fp1);
	}
	fclose(fp);
	fclose(fp1);
	delete[] data;
	hipFree(d_b);
	hipFree(d_a);
	return 0;
}
