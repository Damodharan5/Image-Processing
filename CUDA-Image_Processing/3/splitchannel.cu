#include "hip/hip_runtime.h"
//Split the RGB Channel of 4K image using CUDA
#include "headerinc.h"

#define size_seek(a) ((a) == 24 ? 54:1078)

using namespace std;
typedef struct {
	unsigned char b;
	unsigned char g;
	unsigned char r;
}rgbcolor;
BMPHEADER *_head;
DIBHEADER *_dib;
__global__ void splitchannel(rgbcolor *a, rgbcolor *b, char channel ,unsigned long count)
{

	unsigned long id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < count)
	{

		b[id].r = b[id].g = b[id].b = 0;
		if(channel == 'r' || channel == 'R') b[id].r = a[id].r;
		if(channel == 'g' || channel == 'G') b[id].g = a[id].g;
		if(channel == 'b' || channel == 'B') b[id].b = a[id].b;

	}
}

int main()
{
	FILE *fp, *fp1;
	rgbcolor *d_a,*d_b;
	rgbcolor *data;
	unsigned int colordata[256];
	DWORD pixelarray;
	fp = fopen("C:/Users/cdamo/Desktop/2.bmp", "rb");
	fp1 = fopen("C:/Users/cdamo/Desktop/b2.bmp", "wb");

	_head = (BMPHEADER *)malloc(sizeof(BMPHEADER));
	fread(_head, sizeof(BMPHEADER), 1, fp);
	if (_head->marker != 19778) {
		cout << "Not a bmp file";
	}
	else {
		_dib = (DIBHEADER *)malloc(sizeof(DIBHEADER));
		fread(_dib, sizeof(DIBHEADER), 1, fp);
		if (_dib->bits_pixel == 8) { fread(colordata, 256 * 4, 1, fp); }
		fwrite(_head, 1, sizeof(BMPHEADER), fp1);
		fwrite(_dib, 1, sizeof(DIBHEADER), fp1);
		if (_dib->bits_pixel == 8) fwrite(colordata, 1, 256 * 4, fp1);
		fseek(fp1, size_seek(_dib->bits_pixel), SEEK_SET);
		unsigned int padded = floor((float)(_dib->bits_pixel*_dib->width_pixel + 31.0)/32.0)*4 -_dib->bits_pixel/8*_dib->width_pixel ;
		pixelarray = ((_dib->bits_pixel/8*_dib->width_pixel)+padded)*_dib->height_pixel;
		data = (rgbcolor *)malloc(pixelarray);
		fread(data,pixelarray,1,fp);
		if (hipMalloc(&d_a, pixelarray) != hipSuccess)
		{
		
			cout<<hipGetErrorString(hipGetLastError());
			return 0;
		}
		if (hipMalloc(&d_b, pixelarray) != hipSuccess)
		{

			cout<<hipGetErrorString(hipGetLastError());
			hipFree(d_a);
			return 0;
		}
		if (hipMemcpy(d_a, data, pixelarray, hipMemcpyHostToDevice) != hipSuccess) 
		{

			cout<<hipGetErrorString(hipGetLastError());
			hipFree(d_b);
			hipFree(d_a);
			return 0;
		}
		clock_t begin = clock();
		splitchannel <<<pixelarray / (256*3) + 1 , 256>>> (d_a,d_b,'g',pixelarray);
		clock_t end = clock();
		cout<<((double)end-begin)/CLOCKS_PER_SEC<<" Secs\n";
		if (hipMemcpy(data, d_b, pixelarray, hipMemcpyDeviceToHost) != hipSuccess) 
		{

			cout<<hipGetErrorString(hipGetLastError());
			return 0;
		}
		fwrite(data,1,pixelarray,fp1);
	}
	fclose(fp);
	fclose(fp1);
	delete[] data;
	hipFree(d_b);
	hipFree(d_a);
	return 0;
}
