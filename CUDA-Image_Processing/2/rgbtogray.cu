#include "hip/hip_runtime.h"
#include "headerinc.h"

#define size_seek(a) ((a) == 24 ? 54:1078)

using namespace std;
typedef struct {
	unsigned char r;
	unsigned char g;
	unsigned char b;
}rgbcolor;
BMPHEADER *_head;
DIBHEADER *_dib;
__global__ void Rgbinv(rgbcolor *a, rgbcolor *b, unsigned long count)
{

unsigned long id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < count)
{

	b[id].r = b[id].g = b[id].b = (a[id].b + a[id].g + a[id].r)/3;

}
}

int main()
{
	FILE *fp, *fp1;
	rgbcolor *d_a,*d_b;
	rgbcolor *data;
	unsigned int colordata[256];
	DWORD pixelarray;
	fp = fopen("C:/Users/cdamo/Desktop/1.bmp", "rb");
	fp1 = fopen("C:/Users/cdamo/Desktop/b2.bmp", "wb");

	_head = (BMPHEADER *)malloc(sizeof(BMPHEADER));
	fread(_head, sizeof(BMPHEADER), 1, fp);
	if (_head->marker != 19778) {
		cout << "Not a bmp file";
	}
	else {
		_dib = (DIBHEADER *)malloc(sizeof(DIBHEADER));
		fread(_dib, sizeof(DIBHEADER), 1, fp);
		if (_dib->bits_pixel == 8) { fread(colordata, 256 * 4, 1, fp); }
		fwrite(_head, 1, sizeof(BMPHEADER), fp1);
		fwrite(_dib, 1, sizeof(DIBHEADER), fp1);
		if (_dib->bits_pixel == 8) fwrite(colordata, 1, 256 * 4, fp1);
		fseek(fp1, size_seek(_dib->bits_pixel), SEEK_SET);
		unsigned int padded = floor((float)(_dib->bits_pixel*_dib->width_pixel + 31.0)/32.0)*4 -_dib->bits_pixel/8*_dib->width_pixel ;
		pixelarray = ((_dib->bits_pixel/8*_dib->width_pixel)+padded)*_dib->height_pixel;
		data = (rgbcolor *)malloc(pixelarray);
		fread(data,pixelarray,1,fp);
		if (hipMalloc(&d_a, pixelarray) != hipSuccess)
		{
		
			cout<<hipGetErrorString(hipGetLastError());
			return 0;
		}
		if (hipMalloc(&d_b, pixelarray) != hipSuccess)
		{

			cout<<hipGetErrorString(hipGetLastError());
			hipFree(d_a);
			return 0;
		}
		if (hipMemcpy(d_a, data, pixelarray, hipMemcpyHostToDevice) != hipSuccess) 
		{

			cout<<hipGetErrorString(hipGetLastError());
			hipFree(d_b);
			hipFree(d_a);
			return 0;
		}
		clock_t begin = clock();
		Rgbinv <<<pixelarray / (256*3) + 1 , 256>>> (d_a,d_b,pixelarray);
		clock_t end = clock();
		cout<<((double)end-begin)/CLOCKS_PER_SEC<<" Secs\n";
		if (hipMemcpy(data, d_b, pixelarray, hipMemcpyDeviceToHost) != hipSuccess) 
		{

			cout<<hipGetErrorString(hipGetLastError());
			return 0;
		}
		fwrite(data,1,pixelarray,fp1);
	}
	fclose(fp);
	fclose(fp1);
	delete[] data;
	hipFree(d_b);
	hipFree(d_a);
	return 0;
}
